
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void hello(){
    printf("Blockdim.x:\t%d\n",blockDim.x);
    printf("Blockdim.x:\t%d\n",blockIdx.x);
    printf("Blockdim.x:\t%d\n",threadIdx.x);
    printf("Hello world from device\n");
}

int main(){
    hello<<<1,10>>>();
    printf("Hello world from host\n");
    return 0;
}