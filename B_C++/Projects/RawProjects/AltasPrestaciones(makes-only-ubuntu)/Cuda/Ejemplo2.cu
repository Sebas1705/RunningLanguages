
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"
#include "time.h"

__global__ void suma(int* array1, int* array2, int* result, int size){
    int pos = threadIdx.x+blockIdx.x*blockDim.x;
    result[pos]=array1[pos]+array2[pos];
}

int main(){
    int tam = 10;
    int *a,*b,*c;
    a=(int*)malloc(sizeof(int)*tam);
    b=(int*)malloc(sizeof(int)*tam);
    c=(int*)malloc(sizeof(int)*tam);
    srand(time(NULL));
    for(int i=0;i<tam;i++){
        a[i]=rand()%10000;
        b[i]=rand()%10000;
    }

    //Device:
    int *d_a,*d_b,*d_c;
    hipMalloc(&d_a,sizeof(int)*tam);
    hipMalloc(&d_b,sizeof(int)*tam);
    hipMalloc(&d_c,sizeof(int)*tam);

    hipMemcpy(a,d_a,sizeof(int)*tam,hipMemcpyHostToDevice);
    hipMemcpy(b,d_a,sizeof(int)*tam,hipMemcpyHostToDevice);
    suma<<<1,10>>>(d_a,d_b,d_c,tam);
    hipMemcpy(d_c,c,sizeof(int)*tam,hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for(int i=0;i<10;i++)printf("%d--",a[i]);
    printf("\n");
    for(int i=0;i<10;i++)printf("%d--",b[i]);
    printf("\n");
    for(int i=0;i<10;i++)printf("%d--",c[i]);
    printf("\n");

    free(a);
    free(b);
    free(c);
    
    return 0;
}